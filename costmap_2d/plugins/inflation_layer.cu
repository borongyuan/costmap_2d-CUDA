#include "hip/hip_runtime.h"
#include <costmap_2d/cuda_inflation_layer.h>
#include <cmath>
#include <cstring>
#include <vector>

#define TPB 512

#define NO_INFORMATION 255
#define INSCRIBED_INFLATED_OBSTACLE 253
#define FREE_SPACE 0

using std::abs;
using std::memcpy;
using std::vector;
using costmap_2d::CellData;

__global__ void setCostFloodingInflateUnkown(unsigned char *master, unsigned long master_size_x, unsigned long master_size_y, unsigned char *cachedCost_1D, unsigned int cacheSize, CellData *obstaclesArray, unsigned int obstaclesArray_count, unsigned int inflation_radius, int min_i, int min_j, int max_i, int max_j)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    int temp=id;

    //No need to handle those threads not assigned with tasks
    int obstacleNum=temp/((2*inflation_radius+1)*(2*inflation_radius+1));
    if(obstacleNum>obstaclesArray_count)
        return;
    
    temp%=(2*inflation_radius+1)*(2*inflation_radius+1);

    //No need to handle those threads too far away from obstacles
    int deltay=temp/(2*inflation_radius+1)-inflation_radius;
    int deltax=temp%(2*inflation_radius+1)-inflation_radius;
    if(deltax*deltax+deltay*deltay>inflation_radius*inflation_radius)
        return;

    //No need to handle those threads out of bounds
    int x=obstaclesArray[obstacleNum].src_x_+deltax;
    int y=obstaclesArray[obstacleNum].src_y_+deltay;
    if(x<min_i||x>=max_i||y<min_j||y>=max_j)
        return;
    
    unsigned int index=y*master_size_x+x;
    unsigned char cost=cachedCost_1D[abs(deltay)*cacheSize+abs(deltax)];
    unsigned char old_cost=master[index];

    if(old_cost==NO_INFORMATION&&cost>FREE_SPACE)
        master[index]=cost;
    else
        master[index]=cost>old_cost?cost:old_cost;
}

__global__ void setCostFloodingNoInflateUnkown(unsigned char *master, unsigned long master_size_x, unsigned long master_size_y, unsigned char *cachedCost_1D, unsigned int cacheSize, CellData *obstaclesArray, unsigned int obstaclesArray_count, unsigned int inflation_radius, int min_i, int min_j, int max_i, int max_j)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    int temp=id;

    //No need to handle those threads not assigned with tasks
    int obstacleNum=temp/((2*inflation_radius+1)*(2*inflation_radius+1));
    if(obstacleNum>obstaclesArray_count)
        return;
    
    temp%=(2*inflation_radius+1)*(2*inflation_radius+1);

    //No need to handle those threads too far away from obstacles
    int deltay=temp/(2*inflation_radius+1)-inflation_radius;
    int deltax=temp%(2*inflation_radius+1)-inflation_radius;
    if(deltax*deltax+deltay*deltay>inflation_radius*inflation_radius)
        return;

    //No need to handle those threads out of bounds
    int x=obstaclesArray[obstacleNum].src_x_+deltax;
    int y=obstaclesArray[obstacleNum].src_y_+deltay;
    if(x<min_i||x>=max_i||y<min_j||y>=max_j)
        return;
    
    unsigned int index=y*master_size_x+x;
    unsigned char cost=cachedCost_1D[abs(deltay)*cacheSize+abs(deltax)];
    unsigned char old_cost=master[index];

    if(old_cost==NO_INFORMATION&&cost>INSCRIBED_INFLATED_OBSTACLE)
        master[index]=cost;
    else
        master[index]=cost>old_cost?cost:old_cost;
}

void costmap_2d::cuda::inflation_layer::setCostFlooding(unsigned char *master, unsigned long master_size_x, unsigned long master_size_y, unsigned char **cached_cost, const vector<CellData> &obstacles, unsigned int inflation_radius, bool inflate_unknown, int min_i, int min_j, int max_i, int max_j)
{
    unsigned int cacheSize=inflation_radius+2;
    
    if(obstacles.empty())
        return;
    
    //Compress the original 2D cached cost into 1D for more convenient cuda_memcpy
    unsigned char *cachedCost_1D=NULL;
    hipMallocManaged(&cachedCost_1D, sizeof(unsigned char)*cacheSize*cacheSize, hipMemAttachHost);
    for(int i=0;i<cacheSize;++i)
        memcpy(cachedCost_1D+cacheSize*i,cached_cost[i],cacheSize*sizeof(unsigned char));
    
    CellData *obstaclesArray=NULL;
    hipMallocManaged(&obstaclesArray, sizeof(CellData)*obstacles.size(), hipMemAttachHost);
    memcpy(obstaclesArray,&obstacles[0],obstacles.size()*sizeof(CellData));

    hipStreamAttachMemAsync(NULL, master, 0, hipMemAttachGlobal);
    hipStreamAttachMemAsync(NULL, cachedCost_1D, 0, hipMemAttachGlobal);
    hipStreamAttachMemAsync(NULL, obstaclesArray, 0, hipMemAttachGlobal);

    //2*inflation_radius+1 is actually inflation diameter, but we still had better pass raduis into kernel.
    unsigned long totalWorkload=obstacles.size()*(2*inflation_radius+1)*(2*inflation_radius+1);
    if(inflate_unknown)
        setCostFloodingInflateUnkown<<<(totalWorkload+TPB-1)/TPB,TPB>>>(master,master_size_x,master_size_y,cachedCost_1D,cacheSize,obstaclesArray,obstacles.size(),inflation_radius,min_i,min_j,max_i,max_j);
    else
        setCostFloodingNoInflateUnkown<<<(totalWorkload+TPB-1)/TPB,TPB>>>(master,master_size_x,master_size_y,cachedCost_1D,cacheSize,obstaclesArray,obstacles.size(),inflation_radius,min_i,min_j,max_i,max_j);
    
    hipStreamAttachMemAsync(NULL, master, 0, hipMemAttachHost);
    hipStreamSynchronize(NULL);
    hipFree(cachedCost_1D);
    hipFree(obstaclesArray);
}
