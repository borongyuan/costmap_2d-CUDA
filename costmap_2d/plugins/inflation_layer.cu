#include "hip/hip_runtime.h"
#include <costmap_2d/cuda_inflation_layer.h>
#include <cmath>
#include <cstring>
#include <vector>

#define TPB 512

#define NO_INFORMATION 255
#define INSCRIBED_INFLATED_OBSTACLE 253
#define FREE_SPACE 0

using std::abs;
using std::memcpy;
using std::vector;
using costmap_2d::CellData;

__global__ void setCostFloodingInflateUnkown(unsigned char *master, unsigned long master_size_x, unsigned long master_size_y, unsigned char *cachedCost_1D, unsigned int cacheSize, CellData *obstaclesArray, unsigned int obstaclesArray_count, unsigned int inflation_radius, int min_i, int min_j, int max_i, int max_j)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    int temp=id;

    //No need to handle those threads not assigned with tasks
    int obstacleNum=temp/((2*inflation_radius+1)*(2*inflation_radius+1));
    if(obstacleNum>obstaclesArray_count)
        return;
    
    temp%=(2*inflation_radius+1)*(2*inflation_radius+1);

    //No need to handle those threads too far away from obstacles
    int deltay=temp/(2*inflation_radius+1)-inflation_radius;
    int deltax=temp%(2*inflation_radius+1)-inflation_radius;
    if(deltax*deltax+deltay*deltay>inflation_radius*inflation_radius)
        return;

    //No need to handle those threads out of bounds
    int x=obstaclesArray[obstacleNum].src_x_+deltax;
    int y=obstaclesArray[obstacleNum].src_y_+deltay;
    if(x<min_i||x>=max_i||y<min_j||y>=max_j)
        return;
    
    unsigned int index=y*master_size_x+x;
    unsigned char cost=cachedCost_1D[abs(deltay)*cacheSize+abs(deltax)];
    unsigned char old_cost=master[index];

    if(old_cost==NO_INFORMATION&&cost>FREE_SPACE)
        master[index]=cost;
    else
        master[index]=cost>old_cost?cost:old_cost;
}

__global__ void setCostFloodingNoInflateUnkown(unsigned char *master, unsigned long master_size_x, unsigned long master_size_y, unsigned char *cachedCost_1D, unsigned int cacheSize, CellData *obstaclesArray, unsigned int obstaclesArray_count, unsigned int inflation_radius, int min_i, int min_j, int max_i, int max_j)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    int temp=id;

    //No need to handle those threads not assigned with tasks
    int obstacleNum=temp/((2*inflation_radius+1)*(2*inflation_radius+1));
    if(obstacleNum>obstaclesArray_count)
        return;
    
    temp%=(2*inflation_radius+1)*(2*inflation_radius+1);

    //No need to handle those threads too far away from obstacles
    int deltay=temp/(2*inflation_radius+1)-inflation_radius;
    int deltax=temp%(2*inflation_radius+1)-inflation_radius;
    if(deltax*deltax+deltay*deltay>inflation_radius*inflation_radius)
        return;

    //No need to handle those threads out of bounds
    int x=obstaclesArray[obstacleNum].src_x_+deltax;
    int y=obstaclesArray[obstacleNum].src_y_+deltay;
    if(x<min_i||x>=max_i||y<min_j||y>=max_j)
        return;
    
    unsigned int index=y*master_size_x+x;
    unsigned char cost=cachedCost_1D[abs(deltay)*cacheSize+abs(deltax)];
    unsigned char old_cost=master[index];

    if(old_cost==NO_INFORMATION&&cost>INSCRIBED_INFLATED_OBSTACLE)
        master[index]=cost;
    else
        master[index]=cost>old_cost?cost:old_cost;
}

void costmap_2d::cuda::inflation_layer::setCostFlooding(unsigned char *master, unsigned long master_size_x, unsigned long master_size_y, unsigned char **cached_cost, const vector<CellData> &obstacles, unsigned int inflation_radius, bool inflate_unknown, int min_i, int min_j, int max_i, int max_j)
{
    unsigned int cacheSize=inflation_radius+2;
    
    if(obstacles.empty())
        return;
    
    //Compress the original 2D cached cost into 1D for more convenient cuda_memcpy
    unsigned char *cachedCost_1D=new unsigned char[cacheSize*cacheSize];
    for(int i=0;i<cacheSize;++i)
        memcpy(cachedCost_1D+cacheSize*i,cached_cost[i],cacheSize*sizeof(unsigned char));
    
    CellData *obstaclesArray=new CellData[obstacles.size()];
    memcpy(obstaclesArray,&obstacles[0],obstacles.size()*sizeof(CellData));

    unsigned char *cuda_master;
    unsigned char *cuda_cachedCost_1D;
    CellData *cuda_obstaclesArray;
    hipMalloc(&cuda_master,sizeof(unsigned char)*master_size_x*master_size_y);
    hipMalloc(&cuda_cachedCost_1D,sizeof(unsigned char)*cacheSize*cacheSize);
    hipMalloc(&cuda_obstaclesArray,sizeof(CellData)*obstacles.size());
    
    hipMemcpy(cuda_master,master,sizeof(unsigned char)*master_size_x*master_size_y,hipMemcpyHostToDevice);
    hipMemcpy(cuda_cachedCost_1D,cachedCost_1D,sizeof(unsigned char)*cacheSize*cacheSize,hipMemcpyHostToDevice);
    hipMemcpy(cuda_obstaclesArray,obstaclesArray,sizeof(CellData)*obstacles.size(),hipMemcpyHostToDevice);

    //2*inflation_radius+1 is actually inflation diameter, but we still had better pass raduis into kernel.
    unsigned long totalWorkload=obstacles.size()*(2*inflation_radius+1)*(2*inflation_radius+1);
    if(inflate_unknown)
        setCostFloodingInflateUnkown<<<(totalWorkload+TPB-1)/TPB,TPB>>>(cuda_master,master_size_x,master_size_y,cuda_cachedCost_1D,cacheSize,cuda_obstaclesArray,obstacles.size(),inflation_radius,min_i,min_j,max_i,max_j);
    else
        setCostFloodingNoInflateUnkown<<<(totalWorkload+TPB-1)/TPB,TPB>>>(cuda_master,master_size_x,master_size_y,cuda_cachedCost_1D,cacheSize,cuda_obstaclesArray,obstacles.size(),inflation_radius,min_i,min_j,max_i,max_j);
    
    hipMemcpy(master,cuda_master,sizeof(unsigned char)*master_size_x*master_size_y,hipMemcpyDeviceToHost);
    hipFree(cuda_master);
    hipFree(cuda_cachedCost_1D);
    hipFree(cuda_obstaclesArray);
    delete [] cachedCost_1D;
    delete [] obstaclesArray;
}
