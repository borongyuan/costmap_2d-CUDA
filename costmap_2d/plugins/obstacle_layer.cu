#include "hip/hip_runtime.h"
#include <costmap_2d/obstacle_layer.h>
#include <costmap_2d/cuda_obstacle_layer.h>
#include <costmap_2d/observation.h>

#include <pcl/impl/point_types.hpp>

#include <cmath>

#define TPB 256

using std::max;
using std::ceil;

using pcl::PointXYZ;
using costmap_2d::cuda::obstacle_layer::MyPointXY;

__device__ bool worldToMap(double wx, double wy, unsigned int& mx, unsigned int& my, double origin_x, double origin_y,
    double resolution, unsigned int size_x, unsigned int size_y)
{
  if (wx < origin_x || wy < origin_y)
    return false;

  mx = (int)((wx - origin_x) / resolution);
  my = (int)((wy - origin_y) / resolution);

  if (mx < size_x && my < size_y)
    return true;

  return false;
}

__device__ void bresenham2D(unsigned char *costmap, unsigned char value, unsigned int abs_da, unsigned int abs_db, int error_b, int offset_a,
    int offset_b, unsigned int offset, unsigned int max_length)
{
    unsigned int end = max_length<abs_da?max_length:abs_da;
    for (unsigned int i = 0; i < end; ++i)
    {
        costmap[offset]=value;
        offset += offset_a;
        error_b += abs_db;
        if ((unsigned int)error_b >= abs_da)
        {
            offset += offset_b;
            error_b -= abs_da;
        }
    }
    costmap[offset]=value;
}

__device__ void raytraceLine(unsigned char *costmap, unsigned char value, unsigned int x0, unsigned int y0,
    unsigned int x1, unsigned int y1, unsigned int max_length, unsigned int size_x)
{
    int dx = x1 - x0;
    int dy = y1 - y0;

    unsigned int abs_dx = abs(dx);
    unsigned int abs_dy = abs(dy);

    int offset_dx = dx>0?1:-1;
    int offset_dy = (dy>0?1:-1) * size_x;

    unsigned int offset = y0 * size_x + x0;

    // we need to chose how much to scale our dominant dimension, based on the maximum length of the line
    double dist = hypot((double)dx, (double)dy);
    double scale = (dist == 0.0) ? 1.0 : (1.0<max_length/dist?1.0:max_length/dist);

    // if x is dominant
    if (abs_dx >= abs_dy)
    {
        int error_y = abs_dx / 2;
        bresenham2D(costmap, value, abs_dx, abs_dy, error_y, offset_dx, offset_dy, offset, (unsigned int)(scale * abs_dx));
        return;
    }

    // otherwise y is dominant
    int error_x = abs_dy / 2;
    bresenham2D(costmap, value, abs_dy, abs_dx, error_x, offset_dy, offset_dx, offset, (unsigned int)(scale * abs_dy));
}

__device__ void updateRaytraceBounds(double ox, double oy, double wx, double wy, double range,
    double* min_x, double* min_y, double* max_x, double* max_y)
{
    double dx = wx-ox, dy = wy-oy;
    double full_distance = hypot(dx, dy);
    double scale = 1.0<range/full_distance?1.0:range/full_distance;
    double ex = ox + dx * scale, ey = oy + dy * scale;
    *min_x=min(ex,*min_x);
    *min_y=min(ey,*min_y);
    *max_x=max(ex,*max_x);
    *max_y=max(ey,*max_y);
}

__global__ void rayTraceFreeSpaceKernel(unsigned char *costmap, unsigned char defaultValue, double raytraceRange,
    MyPointXY *cloudArray, int cloudArray_size, double origin_x, double origin_y, double ox, double oy,
    double map_end_x, double map_end_y, double resolution, unsigned int size_x, unsigned int size_y,
    unsigned int x0, unsigned int y0, double *min_x, double *min_y, double *max_x, double *max_y)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if(id>=cloudArray_size)
        return;

    double wx=cloudArray[id].x;
    double wy=cloudArray[id].y;

    // now we also need to make sure that the enpoint we're raytracing
    // to isn't off the costmap and scale if necessary
    double a = wx - ox;
    double b = wy - oy;
    
    // the minimum value to raytrace from is the origin
    if (wx < origin_x)
    {
        double t = (origin_x - ox) / a;
        wx = origin_x;
        wy = oy + b * t;
    }
    if (wy < origin_y)
    {
        double t = (origin_y - oy) / b;
        wx = ox + a * t;
        wy = origin_y;
    }

    // the maximum value to raytrace to is the end of the map
    if (wx > map_end_x)
    {
        double t = (map_end_x - ox) / a;
        wx = map_end_x - .001;
        wy = oy + b * t;
    }
    if (wy > map_end_y)
    {
        double t = (map_end_y - oy) / b;
        wx = ox + a * t;
        wy = map_end_y - .001;
    }

    // now that the vector is scaled correctly... we'll get the map coordinates of its endpoint
    unsigned int x1, y1;

    // check for legality just in case
    if (!worldToMap(wx, wy, x1, y1,origin_x,origin_y,resolution,size_x,size_y))
        return;

    unsigned int cell_raytrace_range = (unsigned int)max(0.0,ceil(raytraceRange/resolution));
    
    // and finally... we can execute our trace to clear obstacles along that line
    raytraceLine(costmap, defaultValue, x0, y0, x1, y1, cell_raytrace_range, size_x);

    updateRaytraceBounds(ox, oy, wx, wy, raytraceRange, min_x, min_y, max_x, max_y);
}

void costmap_2d::cuda::obstacle_layer::rayTraceFreeSpace(unsigned char *costmap, unsigned char defaultValue, double raytraceRange,
    MyPointXY *cloudArray, unsigned int cloudArraySize, double ox, double oy, double origin_x, double origin_y,
    double map_end_x, double map_end_y, double resolution, unsigned int size_x, unsigned int size_y,
    unsigned int x0, unsigned int y0, double *min_x, double *min_y, double *max_x, double *max_y)
{
    MyPointXY *cuda_cloudArray;
    hipMalloc(&cuda_cloudArray,sizeof(MyPointXY)*cloudArraySize);
    hipMemcpy(cuda_cloudArray,cloudArray,sizeof(MyPointXY)*cloudArraySize,hipMemcpyHostToDevice);
    unsigned char *cuda_costmap;
    hipMalloc(&cuda_costmap,sizeof(unsigned char)*size_x*size_y);
    hipMemcpy(cuda_costmap,costmap,sizeof(unsigned char)*size_x*size_y,hipMemcpyHostToDevice);
    double *cuda_min_x,*cuda_min_y,*cuda_max_x,*cuda_max_y;
    hipMalloc(&cuda_min_x,sizeof(double));
    hipMemcpy(cuda_min_x,min_x,sizeof(double),hipMemcpyHostToDevice);
    hipMalloc(&cuda_min_y,sizeof(double));
    hipMemcpy(cuda_min_y,min_y,sizeof(double),hipMemcpyHostToDevice);
    hipMalloc(&cuda_max_x,sizeof(double));
    hipMemcpy(cuda_max_x,max_x,sizeof(double),hipMemcpyHostToDevice);
    hipMalloc(&cuda_max_y,sizeof(double));
    hipMemcpy(cuda_max_y,max_y,sizeof(double),hipMemcpyHostToDevice);

    rayTraceFreeSpaceKernel<<<(cloudArraySize+TPB-1)/TPB,TPB>>>(cuda_costmap,defaultValue,raytraceRange,cuda_cloudArray,cloudArraySize,origin_x,origin_y,ox,oy,map_end_x,map_end_y,resolution,size_x,size_y,x0,y0,cuda_min_x,cuda_min_y,cuda_max_x,cuda_max_y);

    hipMemcpy(costmap,cuda_costmap,sizeof(unsigned char)*size_x*size_y,hipMemcpyDeviceToHost);
    hipMemcpy(min_x,cuda_min_x,sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(min_y,cuda_min_y,sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(max_x,cuda_max_x,sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(max_y,cuda_max_y,sizeof(double),hipMemcpyDeviceToHost);
    hipFree(cuda_cloudArray);
    hipFree(cuda_costmap);
    hipFree(cuda_min_x);
    hipFree(cuda_min_y);
    hipFree(cuda_max_x);
    hipFree(cuda_max_y);
}